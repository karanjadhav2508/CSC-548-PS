#include "hip/hip_runtime.h"
/*
* Single Author info: kjadhav Karan Jadhav
* Group info: NA
* Last modified: 9/17/2017	
*/

#include <stdio.h>
#include <math.h>
#include "mytime.h"

#define THREADS 512
#define MAX_BLOCKS 64

// GPU kernel, we know: THREADS == blockDim.x
__global__ void integrate(int *n, int *blocks, double *gsum) {
  const unsigned int bid = blockDim.x * blockIdx.x + threadIdx.x;
  const unsigned int tid = threadIdx.x;
  double sum;
  int i, arr_len;
  __shared__ double ssum[THREADS];

  //Leibniz implementation
  sum = 0.0;
  for (i = bid; i < *n; i += blockDim.x * *blocks) {
    sum += pow(-1.0,(double)i) / (double)(2*i + 1);
  }
  ssum[tid] = sum * 4.0;
  // block reduction
  __syncthreads();
  //Keeping track of the size of the array we are reducing
  arr_len = blockDim.x;
  for (i = blockDim.x / 2; i > 0; i >>= 1) { /* per block */
    if (tid < i)
      ssum[tid] += ssum[tid + i];
      //handling cases where threads are not a power of two
      //if there is an element(in case of odd sized array) after the one that the last thread strode to, then add that element to the last thread index
      if(tid+1==i && tid+i+1<arr_len) {
	ssum[tid] += ssum[tid+i+1];
      }
    //continue keeping track of the input array on the next reduction iteration
    arr_len = i;
    __syncthreads();
  }
  if (tid == 0)
    gsum[blockIdx.x] = ssum[tid];
}

// number of threads must be a power of 2
__global__ static void global_reduce(int *n, int *blocks, double *gsum)
{
    __shared__ double ssum[THREADS];
    const unsigned int tid = threadIdx.x;
    unsigned int i, arr_len;

    ssum[tid] = gsum[tid];
    __syncthreads();
    arr_len = blockDim.x;
    for (i = blockDim.x / 2; i > 0; i >>= 1) { /* per block */
        if (tid < i)
           ssum[tid] += ssum[tid + i];
           //handling the case where number of blocks is not a power of 2(in global reduction, number of threads is the original number of blocks)
	   //same logic as block reduction earlier
	   if(tid+1==i && tid+i+1<arr_len) {
	      ssum[tid] += ssum[tid+i+1];
	   }
        arr_len = i;
        __syncthreads();
    }
    if (tid == 0)
      gsum[tid] = ssum[tid];
}

int main(int argc, char *argv[]) {
  int n, blocks;
  int *n_d, *blocks_d; // device copy
  double PI25DT = 3.141592653589793238462643;
  double pi;
  double *mypi_d; // device copy of pi
  struct timeval startwtime, endwtime, diffwtime;
  
  // Allocate memory on GPU
  hipMalloc( (void **) &n_d, sizeof(int) * 1 );
  hipMalloc( (void **) &blocks_d, sizeof(int) * 1 );
  hipMalloc( (void **) &mypi_d, sizeof(double) * THREADS * MAX_BLOCKS );

  while (1) {
    printf("Enter the number of intervals: (0 quits) ");fflush(stdout);
    scanf("%d",&n);
    printf("Enter the number of blocks: (<=%d) ", MAX_BLOCKS);fflush(stdout);
    scanf("%d",&blocks);

    gettimeofday(&startwtime, NULL);
    if (n == 0 || blocks > MAX_BLOCKS)
      break;

    // copy from CPU to GPU
    hipMemcpy( n_d, &n, sizeof(int) * 1, hipMemcpyHostToDevice );
    hipMemcpy( blocks_d, &blocks, sizeof(int) * 1, hipMemcpyHostToDevice );

    integrate<<< blocks, THREADS >>>(n_d, blocks_d, mypi_d);
    if (blocks > 1)
      global_reduce<<< 1, blocks >>>(n_d, blocks_d, mypi_d);
    // copy back from GPU to CPU
    hipMemcpy( &pi, mypi_d, sizeof(double) * 1, hipMemcpyDeviceToHost );

    gettimeofday(&endwtime, NULL);
    MINUS_UTIME(diffwtime, endwtime, startwtime);
    printf("pi is approximately %.16f, Error is %.16f\n",
	   pi, fabs(pi - PI25DT));
    printf("wall clock time = %d.%06d\n",
	   diffwtime.tv_sec, diffwtime.tv_usec);
  }

  // free GPU memory
  hipFree(n_d);
  hipFree(blocks_d);
  hipFree(mypi_d);

  return 0;
}
